#include "hip/hip_runtime.h"
#include "ex2.h"

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        std::cout << file << ':' << line << ": [CUDA ERROR] " << hipGetErrorString(code) << std::endl; 
        std::abort();
    }
}

constexpr int T = 16; // threads per bloc


//
// CPU
//
std::vector<int> matvecmul1(
    const std::vector<int>& A,
    const std::vector<int>& b)
{
    // ...
    std::vector<int> res(A.size()/b.size(), 0);

    for(int i =0; i< A.size()/b.size(); i++)
        {
            for(int j = 0; j < b.size(); j++)
            {
                res[i] += A[i * b.size() + j] * b[j];
            }
        }
    return res;
}

namespace kernel {

__global__
void matvecmul2(const int* A, const int* b, int* c, int N, int M)
{
    // ...
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        c[i] = 0;
        for (int j=0; j < M; j++)
        {
            c[i] += A[i*M + j];
        }
    }
}

} // namespace kernel

//
// GPU
//
std::vector<int> matvecmul2(
    const std::vector<int>& A,
    const std::vector<int>& b)
{
    int * dA, *db, *dc;
    CUDA_CHECK(hipMalloc(&dA, A.size()*sizeof(int)));
    CUDA_CHECK(hipMalloc(&db, b.size()*sizeof(int)));
    CUDA_CHECK(hipMalloc(&dc, A.size()/b.size()*sizeof(int)));

    CUDA_CHECK(hipMemcpy(dA, A.data(), A.size()*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(db, b.data(), b.size()*sizeof(int), hipMemcpyHostToDevice));

    kernel::matvecmul2<<<(T + b.size() -1) / T, T>>>(dA, db, dc, A.size()/b.size(), b.size());

    std::vector<int> c;
    CUDA_CHECK(hipMemcpy(c.data(), dc, A.size()/b.size()*sizeof(int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(db));
    CUDA_CHECK(hipFree(dc));
    
    return c;
}

namespace kernel {

__global__
void matvecmul3(const int* A, const int* b, int* c, int N, int M)
{
    // ...
}

} // namespace kernel

//
// GPU by bloc
//
std::vector<int> matvecmul3(
    const std::vector<int>& A,
    const std::vector<int>& b)
{
    // ...
    return {};
}

