
#include <hip/hip_runtime.h>
#include <iostream>

//
// example: CUDA_CHECK( cudaMalloc(dx, x, N*sizeof(int) );
//
#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        std::cout << file << ':' << line << ": [CUDA ERROR] " << hipGetErrorString(code) << std::endl; 
        std::abort();
    }
}

constexpr int bloc_count       = 128; // constexpr equivalent to blockDim.x in CUDA kernel
constexpr int threads_per_bloc = 32;  // constexpr equivalent to gridDim.x  in CUDA kernel

constexpr int B = bloc_count;
constexpr int T = threads_per_bloc;

//
// step 04
//
// dx: array of size N
// dy: array of size N
// dz: array of size B
//

__global__
void dot(int N, const int* dx, const int* dy, int* dz)
{
    __shared__ int buffer[T];

    buffer[threadIdx.x] = 0;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = j; i < N; i += gridDim.x * blockDim.x){
        buffer[threadIdx.x] += dx[i] * dy[i];
    }

    int thread = T / 2;
    while (thread > 1)
    {
        if (threadIdx.x < thread)
        {
            buffer[threadIdx.x] += buffer[threadIdx.x + thread];
            if (blockIdx.x ==0 && threadIdx.x == 0)
                printf("id=%i, thread=%i\n", threadIdx.x, thread);
        }
        thread = thread / 2;
        __syncthreads();
    }

}




int main()
{
    constexpr int N = 1e6;

    int* x = (int*)malloc(N * sizeof(int));
    int* y = (int*)malloc(N * sizeof(int));
    int host_expected_result = 0;
    for (int i = 0; i < N; i++) {
        x[i] = i % 10;
        y[i] = i % 3 - 1;
        host_expected_result += x[i] * y[i];
    }

    // step 05
    int result = 0;
    int *dx, *dy, *dz;
    int *z = (int*)malloc(B * sizeof(int));


    CUDA_CHECK(hipMalloc(&dx, N*sizeof(int)));
    CUDA_CHECK(hipMalloc(&dy, N*sizeof(int)));
    CUDA_CHECK(hipMalloc(&dz, B*sizeof(int)));

    CUDA_CHECK(hipMemcpy(dx, x, N*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dy, y, N*sizeof(int), hipMemcpyHostToDevice));

    dot<<<B,T>>>(N, dx, dy, dz);


    CUDA_CHECK(hipMemcpy(z, dz, B*sizeof(int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(dx));
    CUDA_CHECK(hipFree(dy));
    CUDA_CHECK(hipFree(dz));

    hipDeviceSynchronize();

    for (int i = 0 ; i < B; i++)
    {
        result += z[i];
        printf("z[%i] = %i\n",i, z[i]);
    }



    // checking results
    if(host_expected_result == result) {
        std::cout << "Success" << std::endl;
    } else {
        std::cout << "Error" << std::endl;
        std::cout << "  expected: " << host_expected_result << std::endl;
        std::cout << "  got: " << result << std::endl;
    }

    free(x);
    free(y);
    free(z);

  return 0;
}