
#include <hip/hip_runtime.h>
#include <iostream>


#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

__global__
void kernel1(int* x, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N)
        x[i] *= 2;
}

__global__
void kernel2(int* y, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) 
        y[i] += 1;
}

__global__
void kernel3(const int* x, const int* y, int* z, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) 
        z[i] = x[i] + y[i];
}

__global__
void kernel4(int* z, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) 
        z[i] -= 1;
}

int main(int argc, char const *argv[])
{
    const int T = argc > 1 ? std::stoi(argv[1]) : 512;
    const int N = 4e5;
    const int B = (N+T-1)/T;
    std::cout << "T = " << T << std::endl;
    std::cout << "B = " << B << std::endl;

    int* x = (int*)malloc(N*sizeof(int));
    int* y = (int*)malloc(N*sizeof(int));
    int* z = (int*)malloc(N*sizeof(int));

    for(int i = 0; i < N; ++i)
        x[i] = i;

    for(int i = 0; i < N; ++i)
        y[i] = -i;

    int* dx = nullptr;
    int* dy = nullptr;
    int* dz = nullptr;

    float time;
    hipEvent_t start, stop;
    CUDA_CHECK( hipEventCreate(&start) );
    CUDA_CHECK( hipEventCreate(&stop) );
    CUDA_CHECK( hipEventRecord(start, 0) );

    CUDA_CHECK( hipMalloc(&dx, N*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&dy, N*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&dz, N*sizeof(int)) );
    
    CUDA_CHECK( hipMemcpy(dx, x, N*sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(dy, y, N*sizeof(int), hipMemcpyHostToDevice) );

    kernel1<<<B,T>>>(dx,N);
    CUDA_CHECK( hipGetLastError() );

    kernel2<<<B,T>>>(dy,N);
    CUDA_CHECK( hipGetLastError() );

    kernel3<<<B,T>>>(dx,dy,dz,N);
    CUDA_CHECK( hipGetLastError() );

    kernel4<<<B,T>>>(dz,N);
    CUDA_CHECK( hipGetLastError() );

    CUDA_CHECK( hipMemcpy(z, dz, N*sizeof(int), hipMemcpyDeviceToHost) );

    for(int i = 0; i < N; ++i)
    {
        if(z[i] != i) 
        {
            std::cout << "error at i=" << i << std::endl;
            std::cout << "  expected = " << i << std::endl;
            std::cout << "  got      = " << z[i] << std::endl;
            return 1;
        }
    }

    free(x);
    free(y);
    free(z);
    CUDA_CHECK( hipFree(dx) );
    CUDA_CHECK( hipFree(dy) );
    CUDA_CHECK( hipFree(dz) );

    CUDA_CHECK( hipEventRecord(stop, 0) );
    CUDA_CHECK( hipEventSynchronize(stop) );
    CUDA_CHECK( hipEventElapsedTime(&time, start, stop) );
    std::cout << "time = " << time << std::endl;

    return 0;
}
