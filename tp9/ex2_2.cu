
#include <hip/hip_runtime.h>
#include <iostream>


#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

__global__
void kernel1(int* x, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N)
        x[i] *= 2;
}

__global__
void kernel2(int* y, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) 
        y[i] += 1;
}

__global__
void kernel3(const int* x, const int* y, int* z, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) 
        z[i] = x[i] + y[i];
}

__global__
void kernel4(int* z, int N)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) 
        z[i] -= 1;
}

int main(int argc, char const *argv[])
{
    const int T = argc > 1 ? std::stoi(argv[1]) : 512;
    const int N = 4e5;
    const int B = (N+T-1)/T;
    std::cout << "T = " << T << std::endl;
    std::cout << "B = " << B << std::endl;

    int* x = (int*)malloc(N*sizeof(int));
    int* y = (int*)malloc(N*sizeof(int));
    int* z = (int*)malloc(N*sizeof(int));

    for(int i = 0; i < N; ++i)
        x[i] = i;

    for(int i = 0; i < N; ++i)
        y[i] = -i;

    int* dx = nullptr;
    int* dy = nullptr;
    int* dz = nullptr;


    hipStream_t s1, s2, s3, s4;

    CUDA_CHECK ( hipStreamCreate(&s1) );
    CUDA_CHECK ( hipStreamCreate(&s2) );
    CUDA_CHECK ( hipStreamCreate(&s3) );
    CUDA_CHECK ( hipStreamCreate(&s4) );


    float time;
    hipEvent_t start, stop, e1, e2;
    CUDA_CHECK( hipEventCreate(&start) );
    CUDA_CHECK( hipEventCreate(&stop) );
    CUDA_CHECK( hipEventRecord(start, 0) );

    CUDA_CHECK( hipMalloc(&dx, N*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&dy, N*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&dz, N*sizeof(int)) );
    
    CUDA_CHECK( hipMemcpyAsync(dx, x, N*sizeof(int), hipMemcpyHostToDevice, s1) );
    CUDA_CHECK( hipMemcpyAsync(dy, y, N*sizeof(int), hipMemcpyHostToDevice, s2) );

    kernel1<<<B,T, 0, s1>>>(dx,N);
    CUDA_CHECK( hipEventRecord(e1, 0) );
    CUDA_CHECK( hipGetLastError() );

    kernel2<<<B,T, 0, s2>>>(dy,N);
    CUDA_CHECK( hipEventRecord(e2, 0) );
    CUDA_CHECK( hipGetLastError() );

    CUDA_CHECK( hipStreamWaitEvent(s3, e1));
    CUDA_CHECK( hipStreamWaitEvent(s3, e2));
    CUDA_CHECK( hipStreamWaitEvent(s4, e1));
    CUDA_CHECK( hipStreamWaitEvent(s4, e2));

    kernel3<<<B,T, 0 , s3>>>(dx,dy,dz,N/2);
    CUDA_CHECK( hipGetLastError() );

    kernel3<<<B,T, 0 , s4>>>(dx + N/2, dy + N/2, dz + N/2,N/2);
    CUDA_CHECK( hipGetLastError() );

    kernel4<<<B,T, 0, s4>>>(dz,N/2);
    CUDA_CHECK( hipGetLastError() );

    kernel4<<<B,T, 0, s4>>>(dz + N/2,N/2);
    CUDA_CHECK( hipGetLastError() );

    CUDA_CHECK( hipMemcpyAsync(z, dz, N*sizeof(int)/2, hipMemcpyDeviceToHost, s3) );
    CUDA_CHECK( hipMemcpyAsync(z + N/2, dz+ N/2, N*sizeof(int)/2, hipMemcpyDeviceToHost, s4) );

    for(int i = 0; i < N; ++i)
    {
        if(z[i] != i) 
        {
            std::cout << "error at i=" << i << std::endl;
            std::cout << "  expected = " << i << std::endl;
            std::cout << "  got      = " << z[i] << std::endl;
            return 1;
        }
    }

    free(x);
    free(y);
    free(z);
    CUDA_CHECK( hipFree(dx) );
    CUDA_CHECK( hipFree(dy) );
    CUDA_CHECK( hipFree(dz) );

    CUDA_CHECK( hipEventRecord(stop, 0) );
    CUDA_CHECK( hipEventSynchronize(stop) );
    CUDA_CHECK( hipEventElapsedTime(&time, start, stop) );
    std::cout << "time = " << time << std::endl;

    return 0;
}
