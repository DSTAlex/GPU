
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

namespace kernel {

__global__
void compute(int* x, int N, int iter=100)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N / 4) {
        for(int n = 0; n < iter; ++n)
            x[i] += int(powf(-1,n));
    }
}

} // namespace kernel

//
// simple program that performs dummy computations
//
int main(int argc, char const *argv[])
{
    const int T = argc > 1 ? std::stoi(argv[1]) : 512;
    const int N = 4e8;
    const int B = (N+T-1)/T;
    std::cout << "T = " << T << std::endl;
    std::cout << "B = " << B << std::endl;

    hipStream_t s1, s2, s3, s4;

    CUDA_CHECK ( hipStreamCreate(&s1) );
    CUDA_CHECK ( hipStreamCreate(&s2) );
    CUDA_CHECK ( hipStreamCreate(&s3) );
    CUDA_CHECK ( hipStreamCreate(&s4) );

    hipEvent_t start;
    hipEvent_t stop;

    CUDA_CHECK( hipEventCreate(&start));
    CUDA_CHECK( hipEventCreate(&stop));

    hipEventRecord(start, 0);

    int* x = nullptr;
    CUDA_CHECK( hipHostMalloc(&x, N*sizeof(int), hipHostMallocDefault) );
    for(int i = 0; i < N; ++i)
        x[i] = -N/2 + i;

    int* dx = nullptr;
    CUDA_CHECK( hipMalloc(&dx, N*sizeof(int)) );
    
    // CUDA_CHECK( cudaMemcpy(dx, x, N*sizeof(int), cudaMemcpyHostToDevice) );

    CUDA_CHECK ( hipMemcpyAsync(dx, x, N*sizeof(int) / 4, hipMemcpyHostToDevice, s1) );
    CUDA_CHECK ( hipMemcpyAsync(dx + (N/4), x + (N/4), N*sizeof(int) / 4, hipMemcpyHostToDevice, s2) );
    CUDA_CHECK ( hipMemcpyAsync(dx + 2*(N/4), x + 2*(N/4), N*sizeof(int) / 4, hipMemcpyHostToDevice, s3) );
    CUDA_CHECK ( hipMemcpyAsync(dx + 3*(N/4), x + 3*(N/4), N*sizeof(int) / 4, hipMemcpyHostToDevice, s4) );

    kernel::compute<<<(N/4 + T - 1),T, 0, s1>>>(dx,N/4);
    kernel::compute<<<(N/4 + T - 1),T, 0, s2>>>(dx + (N/4),N/4);
    kernel::compute<<<(N/4 + T - 1),T, 0, s3>>>(dx + 2*(N/4),N/4);
    kernel::compute<<<(N/4 + T - 1),T, 0, s4>>>(dx + 3*(N/4),N/4);
    CUDA_CHECK( hipGetLastError() );

    //CUDA_CHECK( cudaMemcpy(x, dx, N*sizeof(int), cudaMemcpyDeviceToHost) );

    CUDA_CHECK ( hipMemcpyAsync(x, dx, N*sizeof(int) / 4, hipMemcpyDeviceToHost, s1) );
    CUDA_CHECK ( hipMemcpyAsync(x + (N/4), dx + (N/4), N*sizeof(int) / 4, hipMemcpyDeviceToHost, s2) );
    CUDA_CHECK ( hipMemcpyAsync(x + 2*(N/4), dx + 2*(N/4), N*sizeof(int) / 4, hipMemcpyDeviceToHost, s3) );
    CUDA_CHECK ( hipMemcpyAsync(x + 3*(N/4), dx + 3*(N/4), N*sizeof(int) / 4, hipMemcpyDeviceToHost, s4) );

    CUDA_CHECK ( hipStreamSynchronize(s1) );
    CUDA_CHECK ( hipStreamSynchronize(s2) );
    CUDA_CHECK ( hipStreamSynchronize(s3) );
    CUDA_CHECK ( hipStreamSynchronize(s4) );

    CUDA_CHECK (  hipStreamDestroy(s1));
    CUDA_CHECK (  hipStreamDestroy(s2));
    CUDA_CHECK (  hipStreamDestroy(s3));
    CUDA_CHECK (  hipStreamDestroy(s4));

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    printf("temps d'execution: %f ms\n", ms);

    return 0;
}
