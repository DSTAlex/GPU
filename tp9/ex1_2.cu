
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

namespace kernel {

__global__
void compute(int* x, int N, int iter=100)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N) {
        for(int n = 0; n < iter; ++n)
            x[i] += int(powf(-1,n));
    }
}

} // namespace kernel

//
// simple program that performs dummy computations
//
int main(int argc, char const *argv[])
{
    const int T = argc > 1 ? std::stoi(argv[1]) : 512;
    const int N = 4e8;
    const int B = (N+T-1)/T;
    std::cout << "T = " << T << std::endl;
    std::cout << "B = " << B << std::endl;

    hipEvent_t start;
    hipEvent_t stop;

    CUDA_CHECK( hipEventCreate(&start));
    CUDA_CHECK( hipEventCreate(&stop));

    hipEventRecord(start, 0);
    hipEventSynchronize(start);

    int* x = nullptr;
    CUDA_CHECK( hipHostMalloc(&x, N*sizeof(int), hipHostMallocDefault) );
    for(int i = 0; i < N; ++i)
        x[i] = -N/2 + i;

    int* dx = nullptr;
    CUDA_CHECK( hipMalloc(&dx, N*sizeof(int)) );
    CUDA_CHECK( hipMemcpy(dx, x, N*sizeof(int), hipMemcpyHostToDevice) );

    kernel::compute<<<B,T>>>(dx,N);
    CUDA_CHECK( hipGetLastError() );

    CUDA_CHECK( hipMemcpy(x, dx, N*sizeof(int), hipMemcpyDeviceToHost) );


    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    float* ms;
    hipEventElapsedTime(ms, start, stop);

    printf("duree: %f ms\n", ms);

    return 0;
}
