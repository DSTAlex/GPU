
#include <hip/hip_runtime.h>
#include <iostream>

//
// example: CUDA_CHECK( cudaMalloc(dx, x, N*sizeof(int) );
//
#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        std::cout << file << ':' << line << ": [CUDA ERROR] " << hipGetErrorString(code) << std::endl; 
        std::abort();
    }
}


// step 06
__global__
void add_strided(int n, const int *dx, int *dy)
{
     int j = threadIdx.x + blockDim.x * blockIdx.x ;
     for(int i = j; i < n; i += gridDim.x * blockDim.x){
        dy[i] = dx[i] + dy[i];
     }
}




int main()
{
    constexpr int N = 1000;
    int* x = (int*)malloc(N*sizeof(int));
    int* y = (int*)malloc(N*sizeof(int));
    for(int i = 0; i < N; ++i) {
        x[i] = i;
        y[i] = i*i;
    }

    // step 07
    int* dx;
    int* dy;
    // 1. allocate on device
    CUDA_CHECK(hipMalloc(&dx, N*sizeof(int)));
    CUDA_CHECK(hipMalloc(&dy, N*sizeof(int)));



    // 2. copy from host to device

    CUDA_CHECK(hipMemcpy(dx, x, N*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dy, y, N*sizeof(int), hipMemcpyHostToDevice));


    // 3. launch CUDA kernel
    const int threads_per_bloc = 32;
    const int blocs = 8;

    add_strided<<<blocs, threads_per_bloc>>>(N, dx, dy);


    // 4. copy result from device to host
    CUDA_CHECK(hipMemcpy(y, dy, N*sizeof(int), hipMemcpyDeviceToHost));


    // 5. free device memory
    CUDA_CHECK(hipFree(dx));
    CUDA_CHECK(hipFree(dy));


    // checking results
    bool ok = true;
    for(int i = 0; i < N; ++i) {
        const int expected_result = i + i*i;
        if(y[i] != expected_result) {
            std::cout << "Failure" << std::endl;
            std::cout << "Result at index i=" 
                << i << ": expected " 
                << i << '+' << i*i << '=' << expected_result << ", got " << y[i] << std::endl;
            ok = false;
            break;
        }
    }
    if(ok) std::cout << "Success" << std::endl;

    free(x);
    free(y);

    return 0;
}
