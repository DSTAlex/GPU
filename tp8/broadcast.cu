#include "hip/hip_runtime.h"
#include "broadcast.h"

namespace kernel {
    
__global__ void broadcast1(int* x, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i>N)
        return;
    int y = x[i];
    y = __shfl_sync(0xFFFFFFFF, y, 0, N);
    x[i] = y;
}

__global__ void broadcast2(int* x, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i>N)
        return;
    int y = x[i];
    y = __shfl_sync(0xFFFFFFFF, y, 7, 8);
    x[i] = y;

}

__global__ void broadcast3(int* x, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i>N)
        return;
    int y = x[i];
    y = __shfl_sync(0xFFFFFFFF, y, 0, 2);
    x[i] = y;
}

} // namespace kernel