#include "hip/hip_runtime.h"
#include "conv.h"

constexpr int     threads_per_bloc = 16;
constexpr int T = threads_per_bloc;

//
// example: CUDA_CHECK( hipMalloc(dx, x, N*sizeof(int) );
//
#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        std::cout << file << ':' << line << ": [CUDA ERROR] " << hipGetErrorString(code) << std::endl; 
        std::abort();
    }
}


//
// 1D convolution 
// - x: input array of size N
// - y: kernel of odd size M
//
// CPU
//
std::vector<int> conv1(const std::vector<int>& x, const std::vector<int>& y)
{
    //
    // step 01
    //
    const int N = x.size();
    const int M = y.size();
    const int P = (M-1) / 2;

    std::vector<int> z(N);

    for (int i =0 ; i < N ; i++)
    {
        int cov = 0;
        for (int j = 0; j < M; j++)
        {
            int k = i + j - P;
            if (k >= 0 and k < N)
            {
                cov += x[k] * y[j];
            }
        }
        z[i] = cov;
    }


    return z;
}

namespace kernel {

//
// step 02
//
__global__ 
void conv2(const int* dx, const int* dy, int N, int M, int* dz)
{
    const int P = (M-1) / 2;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        int cov = 0;
        for (int j = 0; j < M; j++)
        {
            int k = i + j - P;
            if (k >= 0 and k < N)
            {
                cov += dx[k] * dy[j];
            }
        }
        dz[i] = cov;
    }
}

} // namespace kernel

//
// 1D convolution 
// - x: input array of size N
// - y: kernel of odd size M
//
// GPU (naive)
//
std::vector<int> conv2(const std::vector<int>& x, const std::vector<int>& y)
{
    //
    // step 03
    //
    int N = x.size();
    int M = y.size();
    int *dz, *dx, *dy;

    CUDA_CHECK(hipMalloc(&dz, N*sizeof(int)));
    CUDA_CHECK(hipMalloc(&dx, N*sizeof(int)));
    CUDA_CHECK(hipMalloc(&dy, M*sizeof(int)));
    
    CUDA_CHECK(hipMemcpy(dx, x, N*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dy, y, M*sizeof(int), hipMemcpyHostToDevice));
    
    kernel::conv2(dx, dy, N, M, dz);

    std::vector<int> z(N);

    CUDA_CHECK(hipMemcpy(z.data(), dz, N*sizeof(int), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(dx));
    CUDA_CHECK(hipFree(dy));
    CUDA_CHECK(hipFree(dz));

    return z  
}

namespace kernel {

//
// step 04
//
__global__ 
void conv3(const int* dx, const int* dy, int N, int M, int* dz)
{




}

} // namespace kernel

//
// 1D convolution 
// - x: input array of size N
// - y: kernel of odd size M
//
// GPU (optimized)
//
std::vector<int> conv3(const std::vector<int>& x, const std::vector<int>& y)
{
    //
    // step 05
    //




}
