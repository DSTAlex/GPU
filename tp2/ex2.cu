
#include <hip/hip_runtime.h>
#include <iostream>

//
// example: CUDA_CHECK( cudaMalloc(dx, x, N*sizeof(int) );
//
#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        std::cout << file << ':' << line << ": [CUDA ERROR] " << hipGetErrorString(code) << std::endl; 
        std::abort();
    }
}

//
// step 04
// return a pointer to the value at row i and column j from base_address 
// with pitch in bytes
//
__device__ inline int* get_ptr(int* base_address, int i, int j, size_t pitch) {
    
}

//
// step 05
// CUDA kernel add 
//




int main()
{
    constexpr int rows = 200;
    constexpr int cols = 80;
    int* x = (int*)malloc(rows*cols*sizeof(int));
    int* y = (int*)malloc(rows*cols*sizeof(int));
    for(int i = 0; i < rows*cols; ++i) {
        x[i] = i;
        y[i] = std::pow(-1,i) * i;
    }

    //
    // step 06
    //
    int* dx;
    int* dy;
    size_t pitch;
    // 1. allocate on device

    // 2. copy from host to device

    // 3. launch CUDA kernel
    // const dim3 threads_per_bloc{32,32,1};

    // 4. copy result from device to host

    // 5. free device memory



    // checking results
    bool ok = true;
    for(int i = 0; i < rows*cols; ++i) {
        const int expected_result = std::pow(-1,i) * i + i;
        if(y[i] != expected_result) {
            std::cout << "Failure" << std::endl;
            std::cout << "Result at index i=" 
                << i << ": expected " 
                << std::pow(-1,i) * i << '+' << i << '=' << expected_result << ", got " << y[i] << std::endl;
            ok = false;
            break;
        }
    }
    if(ok) std::cout << "Success" << std::endl;

    free(x);
    free(y);
    
    return 0;
}
